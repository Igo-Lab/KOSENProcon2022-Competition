#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include <algorithm>
#include <iostream>

#include "resolver.h"

#define CUDA_SAFE_CALL(func)                                                                                                  \
    do {                                                                                                                      \
        hipError_t err = (func);                                                                                             \
        if (err != hipSuccess) {                                                                                             \
            fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
            exit(err);                                                                                                        \
        }                                                                                                                     \
    } while (0)

constexpr size_t BASE_AUDIO_N = 88;
constexpr size_t BLOCK_N = 256;

typedef struct {
    uint32_t index;
    uint32_t value;
} pair;

int16_t *srcAudios[BASE_AUDIO_N];  // gpuのメモリポインタ．マルチスレッドで呼び出すときに問題になりそう．
int32_t srclens[BASE_AUDIO_N];
hipStream_t streams[BASE_AUDIO_N];
bool isInit = false;
bool srcLoaded = false;

__global__ void diffSum(const int16_t *__restrict__ chunk, const int16_t *__restrict__ src, uint32_t *sums, const int32_t chunkLen, const int32_t sourceLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = idx + 1;  // 1 start
    if (index >= (chunkLen + sourceLen)) {
        return;
    }
    int clip_starti = max(0, index - sourceLen);
    int clip_endi = min(index, chunkLen);
    int src_starti = max(sourceLen - index, 0);
    int src_endi = min(sourceLen, sourceLen + chunkLen - index);

    unsigned int sum = 0;

    //生problemの前加算
#pragma unroll 8
    for (auto i = 0; i < clip_starti; i++) {
        sum += abs(chunk[i]);
    }

#pragma unroll 8
    for (auto i = clip_starti, j = src_starti; i < clip_endi; i++, j++) {
        sum += abs(chunk[i] - src[j]);
        // sum = __sad(chunk[i], src[j], sum);
    }

    //生problemの後加算
#pragma unroll 8
    for (auto i = clip_endi; i < chunkLen; i++) {
        sum += abs(chunk[i]);
    }

    sums[idx] = sum;
}

__global__ void printest(int16_t *arr, uint32_t len) {
    for (auto i = 0; i < len; i++) {
        printf("%d\n", arr[i]);
    }
}

__global__ void argtest(const int16_t *__restrict__ chunk, const int16_t *__restrict__ src, uint32_t *sums, const int32_t chunkLen, const int32_t sourceLen) {
    printf("chlen: %d, srclen: %d\n", chunkLen, sourceLen);

    for (auto i = 0; i < 10; i++) {
        for (auto j = 0; j < 10; j++) {
            printf("%d ", chunk[i * 10 + j]);
        }
        printf("\n");
    }
    printf("\n");
    for (auto i = 0; i < 10; i++) {
        for (auto j = 0; j < 10; j++) {
            printf("%d ", src[i * 10 + j]);
        }
        printf("\n");
    }
}

//とりあえず何も考えずsrcをコピーして解答領域は都度都度確保することに
// TODO:動確したら直す
void memcpy_src2gpu(const int16_t **srcs, const int32_t *lens) {
    if (!isInit) {
        std::cout << "Didn't be inited. Not processing." << std::endl;
        return;
    }

    for (auto i = 0; i < BASE_AUDIO_N; i++) {
        //メモリ確保
        CUDA_SAFE_CALL(hipMalloc((void **)&(srcAudios[i]), sizeof(int16_t) * lens[i]));
        // 読みデータのコピー
        CUDA_SAFE_CALL(hipMemcpy(srcAudios[i], srcs[i], sizeof(int16_t) * lens[i], hipMemcpyHostToDevice));

        srclens[i] = lens[i];
    }

    srcLoaded = true;
}

// 元読みデータはindex-0スタート．つまり0～87
void resolver(const int16_t *chunk, const int32_t chunk_len, const bool *mask, uint32_t **result_raw) {
    int16_t *chunk_d;
    thrust::device_vector<uint32_t> sum_tmp[BASE_AUDIO_N];
    dim3 block(BLOCK_N);

    pair *result = (pair *)result_raw;

    if (!isInit) {
        std::cout << "Didn't be initialized. Not processing." << std::endl;
        return;
    }

    CUDA_SAFE_CALL(hipMalloc((void **)&chunk_d, sizeof(int16_t) * chunk_len));
    CUDA_SAFE_CALL(hipMemcpy(chunk_d, chunk, sizeof(int16_t) * chunk_len, hipMemcpyHostToDevice));

    for (auto i = 0; i < BASE_AUDIO_N; i++) {
        if (mask[i]) {
            continue;
        }
        sum_tmp[i].resize((chunk_len + srclens[i] - 2));
    }

    for (auto i = 0; i < BASE_AUDIO_N; i++) {
        // もし処理が必要ないならスキップ
        if (mask[i]) {
            continue;
        }

        // if(i == 0){
        //     printf("chlen: %u srclen: %u", chunk_len, srclens[0]);
        //     argtest<<<1, 1>>>(chunk_d, srcAudios[i], thrust::raw_pointer_cast(sum_tmp[0].data()), chunk_len, srclens[0]);
        // }

        dim3 grid(((chunk_len + srclens[i] - 2) + block.x - 1) / block.x);
        printf("srcAudio ID: %d, Block: %d, Grid: %d\n", i + 1, block.x, grid.x);
        printf("sums_d size: %d, %d\n", sum_tmp[i].size(), grid.x * block.x);
        diffSum<<<grid, block, 0, streams[i]>>>(chunk_d, srcAudios[i], thrust::raw_pointer_cast(sum_tmp[i].data()), chunk_len, srclens[i]);
    }

    hipDeviceSynchronize();

    for (auto i = 0; i < BASE_AUDIO_N; i++) {
        // もし処理が必要ないならスキップ
        if (mask[i]) {
            result[i].index = i;
            result[i].value = UINT32_MAX;
            continue;
        }
        std::cout << "dev pass." << i << std::endl;

        // uint32_t itmp = i;
        // uint32_t redtmp = thrust::reduce(
        //     thrust::device,
        //     sum_tmp[i].begin(),
        //     sum_tmp[i].end(),
        //     UINT32_MAX,
        //     thrust::minimum<uint32_t>());

        result[i].index = i;
        result[i].value = thrust::reduce(
            thrust::device,
            sum_tmp[i].begin(),
            sum_tmp[i].end(),
            UINT32_MAX,
            thrust::minimum<uint32_t>());
    }

    hipDeviceSynchronize();

    std::sort(result, result + BASE_AUDIO_N, [](const auto &a, const auto &b) { return a.value < b.value; });

    for (auto j = 0; j < BASE_AUDIO_N; j++) {
        printf("[%u, %u],", result[j].index + 1, result[j].value);
    }

    hipFree(chunk_d);
}

// DLLのロードアンロードにフックしてる
namespace {
struct LoadFook {
    LoadFook() {
        for (auto i = 0; i < BASE_AUDIO_N; i++) {
            hipStreamCreate(&streams[i]);
        }

        isInit = true;
    }

    ~LoadFook() {
        if (!isInit) return;
        for (auto i = 0; i < BASE_AUDIO_N; i++) {
            hipStreamDestroy(streams[i]);
        }
        isInit = false;

        if (srcLoaded) {
            for (auto i = 0; i < BASE_AUDIO_N; i++) {
                hipFree(srcAudios[i]);
            }
        }
    }
} loadfook;
}  // namespace