#include "hip/hip_runtime.h"
#include "add_vector.h"

__global__ void add_vector_dev(float *dest, float *lhs, float *rhs, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        dest[idx] = lhs[idx] + rhs[idx];
    }
}

void add_vector(float *dest, float *lhs, float *rhs, size_t n) {
    float *dev_dest, *dev_lhs, *dev_rhs;
    size_t bytes = n * sizeof(float);

    hipMalloc((void **)&dev_dest, bytes);
    hipMalloc((void **)&dev_lhs, bytes);
    hipMalloc((void **)&dev_rhs, bytes);

    hipMemcpy(dev_lhs, lhs, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_rhs, rhs, bytes, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((n + block.x - 1) / block.x);

    add_vector_dev<<<grid, block>>>(dev_dest, dev_lhs, dev_rhs, n);

    hipMemcpy(dest, dev_dest, bytes, hipMemcpyDeviceToHost);

    hipFree(dev_dest);
    hipFree(dev_lhs);
    hipFree(dev_rhs);
}
